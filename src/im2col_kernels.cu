#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "im2col.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int depth, const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int depth_col, const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
		int d_index = h_index / height_col;
		int d_out = d_index % depth_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int d_in = d_out * stride - pad;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += ((channel_out * depth_col + d_out) * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += ((channel_in * depth + d_in)* height + h_in) * width + w_in;
		for(int k = 0; k < ksize; ++k) {
			for (int i = 0; i < ksize; ++i) {
				for (int j = 0; j < ksize; ++j) {
					int d = d_in + k;
					int h = h_in + i;
					int w = w_in + j;

					*data_col_ptr = (d >=0 && h >= 0 && w >= 0 && h < height && w < width && d < depth) ?
						data_im_ptr[k * height * width + i * width + j] : 0;

					//*data_col_ptr = data_im_ptr[ii * width + jj];

					data_col_ptr += depth_col * height_col * width_col;
				}
			}
		}
    }
}

void im2col_ongpu(float *im,
         int channels, int depth, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int depth_col = (depth + 2 * pad - ksize) / stride + 1;
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * depth_col * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, depth, height, width, ksize, pad,
                stride, depth_col, height_col,
                width_col, data_col);
}
